#include "xmalloc.hpp"

#include <iostream>

void xmallocHost(void** ptr, const size_t count) {
    hipError_t err = hipHostMalloc(ptr, count);
    if (err != hipSuccess) {
        cerr << "xmallocHost error" << endl;
        exit(1);
    }
}


void xmallocDevice(void** ptr, const size_t count) {
    hipError_t err = hipMalloc(ptr, count);
    if (err != hipSuccess) {
        cerr << "xmallocDevice error" << endl;
        exit(1);
    }
}
